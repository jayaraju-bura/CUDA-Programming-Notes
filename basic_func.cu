#include "hip/hip_runtime.h"
%%cuda
#include <stdio.h>
#include <stdlib.h>

__global__ void multiply(int *p, int *q, int*r) {
    *r = (*p) * (*q);
}

int main() {

    int p = 4, q = 5, r = 10;
    int *dev_p, *dev_q, *dev_r;

    int size = sizeof(int);
    hipMalloc((void **)&dev_p, size);
    hipMalloc((void **)&dev_q, size);
    hipMalloc((void **)&dev_r, size);

    hipMemcpy(dev_p, &p, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_q, &q, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_r, &r, size, hipMemcpyHostToDevice);

    multiply <<<1, 1>>> (dev_p, dev_q, dev_r);

    hipError_t err = hipMemcpy(&r, dev_r, size, hipMemcpyDeviceToHost);

    if (err != hipSuccess) {
        printf("CUDA Error copying to host :%s\n", hipGetErrorString(err));
    }

    printf("result is %d ", r);

    hipFree(dev_p);
    hipFree(dev_q);
    hipFree(dev_r);
    return 0;
}
